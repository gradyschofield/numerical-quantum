#include "hip/hip_runtime.h"
#include<hiprand/hiprand_kernel.h>

class BigSum {
    float positiveAccumulators[25];
    float negativeAccumulators[25];
    float orderLimits[25];
    int binOffset;
    int orders;

public:

    BigSum() {
        float minBin = 1E-5;
        orders = 25;
        binOffset = round(log10(minBin));
        double lowestLimit = pow(10, binOffset);
        for (int i = 0; i < orders; ++i) {
            orderLimits[i] = lowestLimit * pow(10, i);
            positiveAccumulators[i] = 0;
            negativeAccumulators[i] = 0;
        }
    }
    void add(float x) {
        int bin = min(orders-1, (int)max(0.0, round(log10(fabs(x))) - binOffset));
        float positiveInc = max(0.0f, x);
        float negativeInc = min(0.0f, x);
        positiveAccumulators[bin] += positiveInc;
        negativeAccumulators[bin] += negativeInc;
    }

    operator double() const {
        double positiveSum = 0;
        double negativeSum = 0;
        for(int i = 0; i < orders; ++i) {
            positiveSum += positiveAccumulators[i];
        }
        for(int i = 0; i < orders; ++i) {
            negativeSum += negativeAccumulators[i];
        }
        return positiveSum + negativeSum;
    }

    void operator+=(double x) {
        add(x);
    }

    void operator-=(double x) {
        add(-x);
    }

    void carryAccumulators() {
        for (int bin = 0; bin < orders - 1; ++bin) {
            float setValue = positiveAccumulators[bin] > orderLimits[bin] ? 0 : positiveAccumulators[bin];
            float carryValue = positiveAccumulators[bin] > orderLimits[bin] ? positiveAccumulators[bin] : 0;
            positiveAccumulators[bin + 1] += carryValue;
            positiveAccumulators[bin] = setValue;
        }
        for (int bin = 0; bin < orders - 1; ++bin) {
            float setValue = negativeAccumulators[bin] < -orderLimits[bin] ? 0 : negativeAccumulators[bin];
            float carryValue = negativeAccumulators[bin] < -orderLimits[bin] ? negativeAccumulators[bin] : 0;
            negativeAccumulators[bin + 1] += carryValue;
            negativeAccumulators[bin] = setValue;
        }
    }

};

extern "C" __global__ void integrate(hiprandState * states,
                                     unsigned long long * seed,
                                     unsigned long long * numSamples,
                                     unsigned long long * inCount,
                                     unsigned long long * outCount) {
    int numThreads = blockDim.x * gridDim.x;
    unsigned long long n = *numSamples / numThreads;
    unsigned long long seq = threadIdx.x + blockIdx.x * blockDim.x;

    hiprandState * state = &states[seq];
    hiprand_init(*seed, seq, 0, state);

    unsigned long long in = 0;
    unsigned long long out = 0;
    for(unsigned long long i = 0; i < n; ++i) {
        float x = 5 * hiprand_uniform(state) - 2;
        float y = 5 * hiprand_uniform(state) - 2;

        float d1 = x*x + y*y;
        float d2 = (x-1)*(x-1) + y*y;
        float d3 = (x-1)*(x-1) + (y-1)*(y-1);
        float d4 = x*x + (y-1)*(y-1);

        int z = d1 < 4 && d2 < 4 && d3 < 4 && d4 < 4 ? 1 : 0;
        in += z;
        out += 1-z;
    }
    inCount[seq] = in;
    outCount[seq] = out;
}