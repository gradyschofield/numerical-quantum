
#include <hip/hip_runtime.h>
#include<hiprand/hiprand_kernel.h>

extern "C" __global__ void integrate(hiprandState * states,
                                     unsigned long long * seed,
                                     unsigned long long * numSamples,
                                     unsigned long long * inCount,
                                     unsigned long long * outCount) {
    int numThreads = blockDim.x * gridDim.x;
    unsigned long long n = *numSamples / numThreads;
    unsigned long long seq = threadIdx.x + blockIdx.x * blockDim.x;

    hiprandState * state = &states[seq];
    hiprand_init(*seed, seq, 0, state);

    unsigned long long in = 0;
    unsigned long long out = 0;
    for(unsigned long long i = 0; i < n; ++i) {
        float x = hiprand_uniform(state) - 0.5f;
        float y = hiprand_uniform(state) - 0.5f;

        int z = x*x + y*y < 0.25f ? 1 : 0;
        in += z;
        out += 1-z;
    }
    inCount[seq] = in;
    outCount[seq] = out;
}