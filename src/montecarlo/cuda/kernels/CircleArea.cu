
#include <hip/hip_runtime.h>
#include<hiprand/hiprand_kernel.h>

extern "C" __global__ void integrate(hiprandState * states,
                                     unsigned long long * seed,
                                     unsigned long long * numSamples,
                                     unsigned long long * inCount,
                                     unsigned long long * outCount) {
    int numThreads = blockDim.x * gridDim.x;
    unsigned long long n = *numSamples / numThreads;
    unsigned long long seq = threadIdx.x + blockIdx.x * blockDim.x;

    hiprandState * state = &states[seq];
    hiprand_init(*seed, seq, 0, state);

    unsigned long long in = 0;
    unsigned long long out = 0;
    for(unsigned long long i = 0; i < n; ++i) {
        float x = 5 * hiprand_uniform(state) - 2;
        float y = 5 * hiprand_uniform(state) - 2;

        float d1 = x*x + y*y;
        float d2 = (x-1)*(x-1) + y*y;
        float d3 = (x-1)*(x-1) + (y-1)*(y-1);
        float d4 = x*x + (y-1)*(y-1);

        int z = d1 < 4 && d2 < 4 && d3 < 4 && d4 < 4 ? 1 : 0;
        in += z;
        out += 1-z;
    }
    inCount[seq] = in;
    outCount[seq] = out;
}