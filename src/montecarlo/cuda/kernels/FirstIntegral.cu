#include "hip/hip_runtime.h"
#include<hiprand/hiprand_kernel.h>

#include<BigSum.cuh>

extern "C" __global__ void firstIntegral(hiprandState * states,
                                         unsigned long long * seed,
                                         unsigned long long * numSamples,
                                         double * totalInVolume,
                                         double * totalIntegral,
                                         double * totalOutVolume) {

    int numThreads = blockDim.x * gridDim.x;
    unsigned long long n = *numSamples / numThreads;
    unsigned long long seq = threadIdx.x + blockIdx.x * blockDim.x;

    hiprandState * state = &states[seq];
    hiprand_init(*seed, seq, 0, state);

    BigSum in, out, integral;
    for(unsigned long long i = 0; i < n; ++i) {

    }
    totalInVolume[seq] = in;
    totalOutVolume[seq] = out;
    totalIntegral[seq] = integral;
}