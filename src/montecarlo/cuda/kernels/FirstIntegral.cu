#include "hip/hip_runtime.h"
#include<hiprand/hiprand_kernel.h>

#include<BigSum.cuh>

extern "C" __global__ void firstIntegral(hiprandState * states,
                                         unsigned long long * seed,
                                         unsigned long long * numSamples,
                                         double * totalInVolume,
                                         double * totalIntegral,
                                         double * totalOutVolume) {

    int numThreads = blockDim.x * gridDim.x;
    unsigned long long n = *numSamples / numThreads;
    unsigned long long seq = threadIdx.x + blockIdx.x * blockDim.x;

    hiprandState * state = &states[seq];
    hiprand_init(*seed, seq, 0, state);

    double in = 0, out = 0, integral = 0;
    for(unsigned long long i = 0; i < n; ++i) {
        float k_r = 2 * hiprand_uniform(state);
        float k_theta = M_PI * hiprand_uniform(state);
        float k_phi = 2 * M_PI * hiprand_uniform(state);

        float q_r = 2 * hiprand_uniform(state);
        float q_theta = M_PI * hiprand_uniform(state);
        float q_phi = 2 * M_PI * hiprand_uniform(state);

        float volumeProduct = k_r * k_r * sin(k_theta) * q_r * q_r * sin(q_theta);

        float k_x = k_r * cos(k_phi) * sin(k_theta);
        float k_y = k_r * sin(k_phi) * sin(k_theta);
        float k_z = k_r * cos(k_theta);

        float q_x = q_r * cos(q_phi) * sin(q_theta);
        float q_y = q_r * sin(q_phi) * sin(q_theta);
        float q_z = q_r * cos(q_theta);

        float sum_x = k_x + q_x;
        float sum_y = k_y + q_y;
        float sum_z = k_z + q_z;
        float sum_len2 = sum_x * sum_x + sum_y * sum_y + sum_z * sum_z;

        out += (k_r > 1 || sum_len2 > 1) ? volumeProduct : 0;
        integral += (k_r <= 1 && sum_len2 <= 1) ? volumeProduct/(q_r*q_r) : 0;
        in += (k_r <= 1 && sum_len2 <= 1) ? volumeProduct : 0;
    }
    totalInVolume[seq] = in;
    totalOutVolume[seq] = out;
    totalIntegral[seq] = integral;
}